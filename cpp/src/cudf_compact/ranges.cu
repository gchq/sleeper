#include "hip/hip_runtime.h"

#include "cudf_compact/ranges.hpp"

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include "cudf_compact/filters.hpp"

#include <iostream>
#include <vector>

namespace gpu_compact::cudf_compact
{

template<typename UnaryFunction>
inline __device__ auto make_counting_transform_iterator(cudf::size_type start, UnaryFunction f) {
    return thrust::make_transform_iterator(thrust::make_counting_iterator(start), f);
}

struct row_total_size
{
    page_info const *cum_pages;
    int const *key_offsets;
    size_t num_keys;

    __device__ inline page_info operator()(page_info const &i) {
        // sum sizes for each input column at this row
        size_t sum = 0;
        // iterate over each global_col_idx
        for (int idx = 0; idx < num_keys; idx++) {
            auto const start = key_offsets[idx];
            auto const end = key_offsets[idx + 1];
            auto iter = make_counting_transform_iterator(0, [&] __device__(int i) { return cum_pages[i].row_count; });
            // for the number of rows in i, find where in the list of pages that number of rows should be inserted
            auto const page_index = thrust::lower_bound(thrust::seq, iter + start, iter + end, i.row_count) - iter;
            // add the size of all those pages to that point for this column to the total size
            sum += cum_pages[page_index].size_bytes;
        }
        // so now we know for the given page i, what the total size of all rows across all files is to that point
        return { i.file_idx, i.rg_idx, i.col_idx, i.page_idx, i.schema_idx, i.global_col_idx, i.row_count, sum };
    }
};

struct page_info_by_index
{
    page_info *data;

    __device__ inline page_info operator()(int index) {
        return data[index];
    }
};

std::deque<scalar_pair> getRanges(std::vector<page_info> const &pages,
  cudf::size_type sort_col,
  parquet::format::Type::type col_type,
  parquet::format::ConvertedType::type conv_type,
  size_t chunk_read_limit,
  std::vector<std::vector<parquet::format::ColumnIndex>> const &indexes_per_file) {
    auto stream = rmm::cuda_stream_default;

    // create page keys (transformed into global column index) and copy to device
    std::vector<int> h_page_keys(pages.size());
    std::transform(
      pages.begin(), pages.end(), h_page_keys.begin(), [](auto const &page) { return page.global_col_idx; });

    // Copy to device
    rmm::device_uvector<int> page_keys(pages.size(), stream);
    hipMemcpyAsync(page_keys.data(), h_page_keys.data(), sizeof(int) * h_page_keys.size(), hipMemcpyDefault, stream);

    // Create device numerical sequence from 0
    rmm::device_uvector<int> page_index(page_keys.size(), stream);
    thrust::sequence(thrust::device, page_index.begin(), page_index.end());

    // Sort key/value with global column index as key and page idx as value. Therefore all pages indexes for a single
    // column are contiguous in page_index. page_keys will now contain contiguous runs of global_col_idxs, e.g. all 0's,
    // followed by all 1's, etc.
    thrust::stable_sort_by_key(
      thrust::device, page_keys.begin(), page_keys.end(), page_index.begin(), thrust::less<int>());

    // copy page_info vector to device
    rmm::device_uvector<page_info> d_pages(pages.size(), stream);
    hipMemcpyAsync(d_pages.data(), pages.data(), sizeof(page_info) * pages.size(), hipMemcpyDefault, stream);

    rmm::device_uvector<page_info> cum_pages(page_keys.size(), stream);
    // Make an iterator of the page indexes that will return the page_info object
    auto page_input = thrust::make_transform_iterator(page_index.begin(), page_info_by_index{ d_pages.data() });

    // Fill cum_pages vector with page_info's where each successive object contains the cumulative row count and data
    // size and order will be file 0, col 0, col 1, ..., col N, file 1 col 0, col 1, ..., col N, ...
    thrust::inclusive_scan_by_key(thrust::device,
      page_keys.begin(),
      page_keys.end(),
      page_input,
      cum_pages.begin(),
      thrust::equal_to{},
      [] __device__(auto const &a, auto const &b) {
          return page_info{ b.file_idx,
              b.rg_idx,
              b.col_idx,
              b.page_idx,
              b.schema_idx,
              b.global_col_idx,
              a.row_count + b.row_count,
              a.size_bytes + b.size_bytes };
      });

    // Now sort that to a new vector by row count
    rmm::device_uvector<page_info> cum_pages_sorted{ cum_pages, stream };
    thrust::sort(thrust::device,
      cum_pages_sorted.begin(),
      cum_pages_sorted.end(),
      [] __device__(page_info const &a, page_info const &b) { return a.row_count < b.row_count; });

    rmm::device_uvector<int> key_offsets(page_keys.size() + 1, stream);
    // Work out how many pages per global_col_idx, e.g. how many pages in file 0 col 0, file 0 col 1, ..., col N,
    // file 1 col 0, file 1 col 1, ...

    // key_offsets_end is iterator positioned at end of filled part of vector
    auto const key_offsets_end = thrust::reduce_by_key(thrust::device,
      page_keys.begin(),
      page_keys.end(),
      thrust::make_constant_iterator(1),
      thrust::make_discard_iterator(),
      key_offsets.begin())
                                   .second;

    // Number of cols * number of files
    size_t const num_unique_keys = key_offsets_end - key_offsets.begin();

    // Reductive sum (first element 0) of key_offsets to get final result, key_offsets gives you index
    // into cum_pages where each new column starts
    thrust::exclusive_scan(thrust::device, key_offsets.begin(), key_offsets.end(), key_offsets.begin());

    // Working from cum_pages_sorted which is sorted based on cumulative row count, create vector of pages with size
    // set to the total size to that row position
    rmm::device_uvector<page_info> aggregated_info(cum_pages.size(), stream);
    thrust::transform(thrust::device,
      cum_pages_sorted.begin(),
      cum_pages_sorted.end(),
      aggregated_info.begin(),
      row_total_size{ cum_pages.data(), key_offsets.data(), num_unique_keys });

    // Just keep the pages for the sorting column
    rmm::device_uvector<page_info> d_filtered_pages(aggregated_info.size(), stream);
    auto filtered_end = thrust::copy_if(thrust::device,
      aggregated_info.begin(),
      aggregated_info.end(),
      d_filtered_pages.begin(),
      [sort_col] __device__(auto const &pg) { return pg.schema_idx == sort_col; });
    d_filtered_pages.resize(std::distance(d_filtered_pages.begin(), filtered_end), stream);

    // bring filtered_pages to host for last step
    std::vector<page_info> filtered_pages(d_filtered_pages.size());
    hipMemcpyAsync(filtered_pages.data(),
      d_filtered_pages.data(),
      sizeof(page_info) * d_filtered_pages.size(),
      hipMemcpyDefault,
      stream);

    // wait for all pending operations
    stream.synchronize();

    std::deque<scalar_pair> ranges;
    size_t cur_pos = 0;
    size_t cur_cumulative_size = 0;
    size_t cur_row_count = 0;
    auto start = thrust::make_transform_iterator(
      filtered_pages.begin(), [&](page_info const &i) { return i.size_bytes - cur_cumulative_size; });
    auto end = start + filtered_pages.size();

    auto last_scalar = min_for_type(col_type, conv_type);
    auto max_scalar = max_for_type(col_type, conv_type);

    std::string last_val = "-inf";
    std::string const max_val = "inf";
    while (true) {
        int64_t split_pos = thrust::lower_bound(thrust::seq, start + cur_pos, end, chunk_read_limit) - start;

        // if we're past the end, or if the returned bucket is > than the chunk_read_limit, move
        // back one.
        if (static_cast<size_t>(split_pos) >= filtered_pages.size()
            || (filtered_pages[split_pos].size_bytes - cur_cumulative_size > chunk_read_limit)) {
            split_pos--;
        }

        // best-try. if we can't find something that'll fit, we have to go bigger. we're doing
        // this in a loop because all of the cumulative sizes for all the pages are sorted into
        // one big list. so if we had two columns, both of which had an entry {1000, 10000},
        // that entry would be in the list twice. so we have to iterate until we skip past all
        // of them.  The idea is that we either do this, or we have to call unique() on the
        // input first.
        while (split_pos < (static_cast<int64_t>(filtered_pages.size()) - 1)
               && (split_pos < 0 || filtered_pages[split_pos].row_count == cur_row_count)) {
            split_pos++;
        }
        auto const start_row = cur_row_count;
        cur_row_count = filtered_pages[split_pos].row_count;

        if (cur_row_count == start_row)
            break;

        cur_pos = split_pos;
        cur_cumulative_size = filtered_pages[split_pos].size_bytes;

        auto const &splt = filtered_pages[split_pos];
        auto const &colidx = indexes_per_file[splt.file_idx][splt.col_idx];
        auto const &min = colidx.min_values[splt.page_idx];
        auto const &end_val = split_pos == filtered_pages.size() - 1 ? max_val : min;
        auto end_sclr = to_scalar(min, col_type, conv_type);

        ranges.emplace_back(scalar_pair{ to_string(last_val, col_type, conv_type),
          last_scalar,
          to_string(end_val, col_type, conv_type),
          split_pos == filtered_pages.size() - 1 ? max_scalar : end_sclr });

        std::string rangeBegin = to_string(last_val, col_type, conv_type);
        std::string rangeEnd = to_string(end_val, col_type, conv_type);

        std::cout << "Adding range \"" << rangeBegin << "\"->\"" << rangeEnd << "\"\n";

        last_val = min;
        last_scalar = end_sclr;
    }

    return ranges;
}

}// namespace gpu_compact::cudf_compact